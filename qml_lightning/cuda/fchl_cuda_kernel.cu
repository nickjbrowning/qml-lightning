#include "hip/hip_runtime.h"
#include <math.h>
#include<torch/torch.h>
#include <iostream>

using namespace std;

#define GAUSSIAN_DISTRIBUTION 0
#define LOGNORMAL_DISTRIBUTION 1
#define EXPEXP_DISTRIBUTION 2

#define COSINE_CUTOFF 0
#define SWITCH_FUNCTION 1

#define SQRT2PI 2.506628275f

__device__ void get_pbc_drij(float *drij, float *cell_vectors, float *inv_cell_vectors) {

	/*
	 *   h := [a, b, c], a=(a1,a2,a3), ... (the matrix of box vectors)
	 r_ij := r_i - r_j                 (difference vector)

	 s_i = h^{-1} r_i
	 s_ij = s_i - s_j
	 s_ij <-- s_ij - NINT(s_ij)        (general minimum image convention)
	 r_ij = h s_ij
	 */
	float sij[3] = { 0.0, 0.0, 0.0 };

	for (int m = 0; m < 3; m++) {

		for (int k = 0; k < 3; k++) {

			sij[m] += inv_cell_vectors[m * 3 + k] * drij[k];
		}

		sij[m] = sij[m] - round(sij[m]);
	}

	for (int m = 0; m < 3; m++) {

		float rij_m = 0.0;

		for (int k = 0; k < 3; k++) {
			rij_m += cell_vectors[m * 3 + k] * sij[k];
		}

		drij[m] = rij_m;
	}
}

__device__ float get_cutoff(float rij, float rcut, float rswitch, int cutoff_type) {
	float cut = 1.0;

	switch (cutoff_type) {

	case COSINE_CUTOFF:
		cut = 0.5 * (cosf(rij * M_PI / rcut) + 1.0);
		break;

	case SWITCH_FUNCTION:
		if (rij > rswitch) {
			float sx = (rij - rswitch) / (rcut - rswitch);
			cut = cut - 6.0 * powf(sx, 5.0) + 15.0 * powf(sx, 4.0) - 10.0 * powf(sx, 3.0);
		}
		break;
	default:
		cut = 0.5 * (cosf(rij * M_PI / rcut) + 1.0);
		break;
	}

	return cut;
}

__device__ float get_cutoff_derivative(float rij, float rcut, float rswitch, int cutoff_type) {
	float dcut = 0.0;

	switch (cutoff_type) {

	case COSINE_CUTOFF:

		dcut = -0.5 * (sinf(rij * M_PI / rcut)) * M_PI / rcut;
		break;

	case SWITCH_FUNCTION:

		if (rij > rswitch) {
			float sx = (rij - rswitch) / (rcut - rswitch);
			dcut = (1.0 / (rcut - rswitch)) * (-30.0 * powf(sx, 4.0) + 60.0 * powf(sx, 3.0) - 30.0 * powf(sx, 2.0));
		}
		break;

	default:
		dcut = -0.5 * (sinf(rij * M_PI / rcut)) * M_PI / rcut;
		break;
	}

	return dcut;
}

__device__ float get_radial_distribution(float rij, float eta, float *gridpoints, int index, int distribution_type) {

	float d = 0.0;
	float mu = 0.0;
	float sigma2 = 0.0;
	float sigma = 0.0;

	switch (distribution_type) {

	case GAUSSIAN_DISTRIBUTION:
		d = sqrt(eta / M_PI) * expf(-eta * powf(rij - gridpoints[index], 2.0));
		break;

	case LOGNORMAL_DISTRIBUTION:
		mu = log(rij / sqrt(1.0 + eta / powf(rij, 2.0)));
		sigma2 = log(1.0 + eta / powf(rij, 2.0));
		sigma = sqrt(sigma2);

		d = 1.0 / (gridpoints[index] * sigma * SQRT2PI) * expf(-powf(log(gridpoints[index]) - mu, 2.0) / (2.0 * sigma2));
		break;

	case EXPEXP_DISTRIBUTION:
		d = expf(-eta * powf(expf(-rij) - gridpoints[index], 2.0));
		break;

	default:
		d = sqrt(eta / M_PI) * expf(-eta * powf(rij - gridpoints[index], 2.0));
		break;
	}

	return d;
}

__device__ float get_radial_derivative_distribution(float drijx, float rij, float eta, float *gridpoints, int index, int distribution_type) {

	float dradial_dx = 0.0;

	float sqrt_eta = sqrt(eta / M_PI);
	float mu = 0.0;
	float sigma = 0.0;
	float sigma2 = 0.0;
	float sigma4 = 0.0;
	float lnRs = 0.0;
	float exp_ln = 0.0;
	float rij2 = 0.0;
	float dmu_dx = 0.0;
	float dsigma_dx = 0.0;

	switch (distribution_type) {

	case GAUSSIAN_DISTRIBUTION:
		dradial_dx = sqrt_eta * expf(-eta * powf(rij - gridpoints[index], 2.0)) * -eta * 2.0 * (rij - gridpoints[index]) * -drijx;
		break;

	case LOGNORMAL_DISTRIBUTION:

		mu = log(rij / sqrt(1.0 + (eta / powf(rij, 2.0))));
		sigma = sqrt(log(1.0 + (eta / powf(rij, 2.0))));
		sigma2 = powf(sigma, 2.0);
		sigma4 = powf(sigma, 4.0);
		lnRs = log(gridpoints[index]);
		exp_ln = expf(-powf(lnRs - mu, 2.0) / powf(sigma, 2.0) * 0.5);
		rij2 = powf(rij, 2.0);

		dsigma_dx = drijx * eta * (1.0 / ((eta + rij2) * rij * sqrt(log((eta + rij2) / rij2))));
		dmu_dx = -drijx * ((2 * eta + rij2) / ((eta + rij2) * rij));

		dradial_dx = (sqrt(2.0) / (2 * sqrt(M_PI) * gridpoints[index] * sigma4))
				* (((mu - lnRs) * dsigma_dx - sigma * dmu_dx) * (mu - lnRs) - sigma2 * dsigma_dx) * exp_ln;

		break;

	case EXPEXP_DISTRIBUTION:
		dradial_dx = 2.0 * eta * (-gridpoints[index] + expf(-rij)) * expf(-eta * powf(expf(-rij) - gridpoints[index], 2.0)) * expf(-rij) * -drijx;
		break;

	default:
		dradial_dx = sqrt_eta * expf(-eta * powf(rij - gridpoints[index], 2.0)) * -eta * 2.0 * (rij - gridpoints[index]) * -drijx;
		break;
	}

	return dradial_dx;
}

__device__ float dot_abcd(float *ab, float *cd) {
	return ab[0] * cd[0] + ab[1] * cd[1] + ab[2] * cd[2];
}

__device__ float dot(float *v1, float *v2, float *v3, float *v4) {
	return (v1[0] - v2[0]) * (v3[0] - v4[0]) + (v1[1] - v2[1]) * (v3[1] - v4[1]) + (v1[2] - v2[2]) * (v3[2] - v4[2]);
}

//			float cos_1 = calc_cos_angle(rj, ri, rk); // ji, ki
//			float cos_2 = calc_cos_angle(rj, rk, ri); // jk, ik
//			float cos_3 = calc_cos_angle(ri, rj, rk); // ij, kj

__device__ float calc_cos_angle_abcb(float *ab, float *cb) {

	float v1norm = sqrt(ab[0] * ab[0] + ab[1] * ab[1] + ab[2] * ab[2]);
	float v2norm = sqrt(cb[0] * cb[0] + cb[1] * cb[1] + cb[2] * cb[2]);

	float v1[3];
	float v2[3];

	v1[0] = ab[0] / v1norm;
	v1[1] = ab[1] / v1norm;
	v1[2] = ab[2] / v1norm;

	v2[0] = cb[0] / v2norm;
	v2[1] = cb[1] / v2norm;
	v2[2] = cb[2] / v2norm;

	float cos_angle = v1[0] * v2[0] + v1[1] * v2[1] + v1[2] * v2[2]; //v1.dot(v2);

	return cos_angle;
}

__device__ float calc_cos_angle(float *a, float *b, float *c) {

	float v1norm = sqrt((a[0] - b[0]) * (a[0] - b[0]) + (a[1] - b[1]) * (a[1] - b[1]) + (a[2] - b[2]) * (a[2] - b[2]));
	float v2norm = sqrt((c[0] - b[0]) * (c[0] - b[0]) + (c[1] - b[1]) * (c[1] - b[1]) + (c[2] - b[2]) * (c[2] - b[2]));

	float v1[3];
	float v2[3];

	v1[0] = (a[0] - b[0]) / v1norm;
	v1[1] = (a[1] - b[1]) / v1norm;
	v1[2] = (a[2] - b[2]) / v1norm;

	v2[0] = (c[0] - b[0]) / v2norm;
	v2[1] = (c[1] - b[1]) / v2norm;
	v2[2] = (c[2] - b[2]) / v2norm;

	float cos_angle = v1[0] * v2[0] + v1[1] * v2[1] + v1[2] * v2[2]; //v1.dot(v2);

	return cos_angle;
}

__device__ float calc_angle(float *a, float *b, float *c) {

	float cos_angle = calc_cos_angle(a, b, c);

	if (cos_angle > 1.0)
		cos_angle = 1.0;
	if (cos_angle < -1.0)
		cos_angle = -1.0;

	return acosf(cos_angle);

}

__device__ float calc_angle_abcb(float *ab, float *cb) {

	float cos_angle = calc_cos_angle_abcb(ab, cb);

	if (cos_angle > 1.0)
		cos_angle = 1.0;
	if (cos_angle < -1.0)
		cos_angle = -1.0;

	return acosf(cos_angle);

}

__global__ void fchl19_representation_cuda_old(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> charges,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> species,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> element_types,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> cell,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_cell,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockAtomIDs, // blockIdx -> atom idx
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockMolIDs, // blockIdx -> molecule jdx
		const torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> neighbourlist,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> nneighbours, const int max_neighbours,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs2,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight,
		float three_body_decay, float rcut, torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> output) {

	extern __shared__ int s[];

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);

	int nelements = species.size(0);

	float *scoords_x = (float*) &s;
	float *scoords_y = (float*) &scoords_x[max_neighbours];
	float *scoords_z = (float*) &scoords_y[max_neighbours];
	int *selement_types = (int*) &scoords_z[max_neighbours];

	float *sRs2 = (float*) &selement_types[max_neighbours];
	float *sRs3 = (float*) &sRs2[nRs2];

	float *scell = (float*) &sRs3[nRs3];
	float *sinv_cell = (float*) &scell[9];

	int molID = blockMolIDs[blockIdx.x];
	int iatom = blockAtomIDs[blockIdx.x];

	int repsize = species.size(0) * nRs2 + (species.size(0) * (species.size(0) + 1)) * nRs3;

	__syncthreads();

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nRs2; i += blockDim.x * blockDim.y) {
		sRs2[i] = Rs2[i];
	}
	__syncthreads();

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nRs3; i += blockDim.x * blockDim.y) {
		sRs3[i] = Rs3[i];
	}

	__syncthreads();

	int nneighbours_i = nneighbours[molID][iatom];

	for (int jatom = threadIdx.y * blockDim.x + threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x * blockDim.y) {

		int j = neighbourlist[molID][iatom][jatom];

		scoords_x[jatom] = coordinates[molID][j][0];
		scoords_y[jatom] = coordinates[molID][j][1];
		scoords_z[jatom] = coordinates[molID][j][2];
		selement_types[jatom] = element_types[molID][j];

	}
	__syncthreads();

	float ri[3];
	float rj[3];
	float rk[3];

	float drij[3];
	float drik[3];
	float drjk[3];
	float drji[3];
	float drki[3];
	float drkj[3];

	bool pbc = false;

	if (cell.size(0) > 0) {

		pbc = true;

		if (threadIdx.x == 0 && threadIdx.y == 0) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					scell[i * 3 + j] = cell[molID][i][j];
					sinv_cell[i * 3 + j] = inv_cell[molID][i][j];
				}
			}
		}
	}
	__syncthreads();

	ri[0] = coordinates[molID][iatom][0];
	ri[1] = coordinates[molID][iatom][1];
	ri[2] = coordinates[molID][iatom][2];

	float expf_v = expf(-powf(M_PI, 2) * 0.5);
	float sqrt2pi = sqrtf(2.0 * M_PI);

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		rj[0] = scoords_x[jatom];
		rj[1] = scoords_y[jatom];
		rj[2] = scoords_z[jatom];

		int jelement = selement_types[jatom];

		drij[0] = ri[0] - rj[0];
		drij[1] = ri[1] - rj[1];
		drij[2] = ri[2] - rj[2];

		if (pbc) {
			get_pbc_drij(drij, scell, sinv_cell);
		}

		drji[0] = -drij[0];
		drji[1] = -drij[1];
		drji[2] = -drij[2];

		float rij = sqrtf(drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2]);

		float scaling = 1.0 / powf(rij, two_body_decay);

		float rcutij = get_cutoff(rij, rcut, 0.0, 0);

		float mu = log(rij / sqrt(1.0 + eta2 / powf(rij, 2.0)));
		float sigma = sqrtf(log(1.0 + eta2 / powf(rij, 2.0)));

		float invsigma22 = 1.0 / (2.0 * powf(sigma, 2));

		for (int z = 0; z < nRs2; z++) {

			float radial = 1.0 / (sigma * sqrt2pi * sRs2[z]) * expf(-powf(log(sRs2[z]) - mu, 2) * invsigma22) * scaling * rcutij;

			atomicAdd(&output[molID][iatom][jelement * nRs2 + z], radial);

		}

		for (int katom = jatom + 1 + threadIdx.y; katom < nneighbours_i; katom += blockDim.y) {

			rk[0] = scoords_x[katom];
			rk[1] = scoords_y[katom];
			rk[2] = scoords_z[katom];

			int kelement = selement_types[katom];

			drik[0] = ri[0] - rk[0];
			drik[1] = ri[1] - rk[1];
			drik[2] = ri[2] - rk[2];

			if (pbc) {
				get_pbc_drij(drik, scell, sinv_cell);
			}
			drjk[0] = drik[0] - drij[0];
			drjk[1] = drik[1] - drij[1];
			drjk[2] = drik[2] - drij[2];

			drki[0] = -drik[0];
			drki[1] = -drik[1];
			drki[2] = -drik[2];

			drkj[0] = -drjk[0];
			drkj[1] = -drjk[1];
			drkj[2] = -drjk[2];

			float rik = sqrt(drik[0] * drik[0] + drik[1] * drik[1] + drik[2] * drik[2]);

			if (rik > rcut) {
				continue;
			}

			float rjk = sqrt(drjk[0] * drjk[0] + drjk[1] * drjk[1] + drjk[2] * drjk[2]);

			float rcutik = get_cutoff(rik, rcut, 0.0, 0);

			//__device__ float calc_cos_angle(float *a, float *b, float *c) {
			//float v1norm = sqrt((a[0] - b[0]) * (a[0] - b[0]) + (a[1] - b[1]) * (a[1] - b[1]) + (a[2] - b[2]) * (a[2] - b[2]));
			//float v2norm = sqrt((c[0] - b[0]) * (c[0] - b[0]) + (c[1] - b[1]) * (c[1] - b[1]) + (c[2] - b[2]) * (c[2] - b[2]));

			//float angle = calc_angle(rj, ri, rk);
			float angle = calc_angle_abcb(drji, drki);

			//float cos_1 = calc_cos_angle(rj, ri, rk); // ji, ki
			float cos_1 = calc_cos_angle_abcb(drji, drki); // ji, ki
			//float cos_2 = calc_cos_angle(rj, rk, ri); // jk, ik
			float cos_2 = calc_cos_angle_abcb(drjk, drik); // jk, ik
			//float cos_3 = calc_cos_angle(ri, rj, rk); // ij, kj
			float cos_3 = calc_cos_angle_abcb(drij, drkj); // ij, kj

			float ksi3 = three_body_weight * (1.0 + 3 * cos_1 * cos_2 * cos_3) / powf(rij * rik * rjk, three_body_decay);

			float cos_angle = expf_v * 2.0 * cosf(angle);
			float sin_angle = expf_v * 2.0 * sinf(angle);

			int p = min(jelement, kelement);
			int q = max(jelement, kelement);

			int s = nelements * nRs2 + nRs3 * 2 * (-(p * (p + 1)) / 2 + q + nelements * p);

			for (int l = 0; l < nRs3; l++) {

				int z = s + l * 2;

				float radial = expf(-eta3 * powf(0.5 * (rij + rik) - sRs3[l], 2.0)) * rcutik * rcutij;

				atomicAdd(&output[molID][iatom][z], radial * cos_angle * ksi3);
				atomicAdd(&output[molID][iatom][z + 1], radial * sin_angle * ksi3);
			}
		}
	}
}

__global__ void fchl19_representation_cuda(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> charges,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> species,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> element_types,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> cell,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_cell,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockAtomIDs, // blockIdx -> atom idx
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockMolIDs, // blockIdx -> molecule jdx
		const torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> neighbourlist,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> nneighbours, const int max_neighbours,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs2,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight,
		float three_body_decay, float rcut, torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> output) {

	extern __shared__ int s[];

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);

	int nelements = species.size(0);

	float *scoords_x = (float*) &s;
	float *scoords_y = (float*) &scoords_x[max_neighbours];
	float *scoords_z = (float*) &scoords_y[max_neighbours];
	int *selement_types = (int*) &scoords_z[max_neighbours];

	float *sRs2 = (float*) &selement_types[max_neighbours];
	float *sRs3 = (float*) &sRs2[nRs2];

	float *scell = (float*) &sRs3[nRs3];
	float *sinv_cell = (float*) &scell[9];

	int molID = blockMolIDs[blockIdx.x];
	int iatom = blockAtomIDs[blockIdx.x];

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nRs2; i += blockDim.x * blockDim.y) {
		sRs2[i] = Rs2[i];
	}
	__syncthreads();

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nRs3; i += blockDim.x * blockDim.y) {
		sRs3[i] = Rs3[i];
	}

	__syncthreads();

	int nneighbours_i = nneighbours[molID][iatom];

	for (int jatom = threadIdx.y * blockDim.x + threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x * blockDim.y) {

		int j = neighbourlist[molID][iatom][jatom];

		scoords_x[jatom] = coordinates[molID][j][0];
		scoords_y[jatom] = coordinates[molID][j][1];
		scoords_z[jatom] = coordinates[molID][j][2];
		selement_types[jatom] = element_types[molID][j];

	}

	__syncthreads();

	float ri[3];

	float drij[3];
	float drik[3];
	float drjk[3];
	float drji[3];
	float drki[3];
	float drkj[3];

	bool pbc = false;

	if (cell.size(0) > 0) {

		pbc = true;

		if (threadIdx.x == 0 && threadIdx.y == 0) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					scell[i * 3 + j] = cell[molID][i][j];
					sinv_cell[i * 3 + j] = inv_cell[molID][i][j];
				}
			}
		}
	}
	__syncthreads();

	ri[0] = coordinates[molID][iatom][0];
	ri[1] = coordinates[molID][iatom][1];
	ri[2] = coordinates[molID][iatom][2];

	float expf_v = expf(-powf(M_PI, 2) * 0.5);
	float sqrt2pi = sqrtf(2.0 * M_PI);

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		int jelement = selement_types[jatom];

		drij[0] = ri[0] - scoords_x[jatom];
		drij[1] = ri[1] - scoords_y[jatom];
		drij[2] = ri[2] - scoords_z[jatom];

		if (pbc) {
			get_pbc_drij(drij, scell, sinv_cell);
		}

		drji[0] = -drij[0];
		drji[1] = -drij[1];
		drji[2] = -drij[2];

		float rij = sqrtf(drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2]);

		float scaling = 1.0 / powf(rij, two_body_decay);

		float rcutij = get_cutoff(rij, rcut, 0.0, 0);

		float mu = log(rij / sqrt(1.0 + eta2 / powf(rij, 2.0)));
		float sigma = sqrtf(log(1.0 + eta2 / powf(rij, 2.0)));

		float invsigma22 = 1.0 / (2.0 * powf(sigma, 2));

		for (int z = threadIdx.y; z < nRs2; z += blockDim.y) {

			float radial = 1.0 / (sigma * sqrt2pi * sRs2[z]) * expf(-powf(log(sRs2[z]) - mu, 2) * invsigma22) * scaling * rcutij;

			atomicAdd(&output[molID][iatom][jelement * nRs2 + z], radial);

		}

		for (int katom = jatom + 1 + threadIdx.y; katom < nneighbours_i; katom += blockDim.y) {

			int kelement = selement_types[katom];

			drik[0] = ri[0] - scoords_x[katom];
			drik[1] = ri[1] - scoords_y[katom];
			drik[2] = ri[2] - scoords_z[katom];

			if (pbc) {
				get_pbc_drij(drik, scell, sinv_cell);
			}

			drjk[0] = drik[0] - drij[0];
			drjk[1] = drik[1] - drij[1];
			drjk[2] = drik[2] - drij[2];

			drki[0] = -drik[0];
			drki[1] = -drik[1];
			drki[2] = -drik[2];

			drkj[0] = -drjk[0];
			drkj[1] = -drjk[1];
			drkj[2] = -drjk[2];

			float rik = sqrt(drik[0] * drik[0] + drik[1] * drik[1] + drik[2] * drik[2]);

			if (rik > rcut) {
				continue;
			}

			float rjk = sqrt(drjk[0] * drjk[0] + drjk[1] * drjk[1] + drjk[2] * drjk[2]);

			float rcutik = get_cutoff(rik, rcut, 0.0, 0);

			//__device__ float calc_cos_angle(float *a, float *b, float *c) {
			//float v1norm = sqrt((a[0] - b[0]) * (a[0] - b[0]) + (a[1] - b[1]) * (a[1] - b[1]) + (a[2] - b[2]) * (a[2] - b[2]));
			//float v2norm = sqrt((c[0] - b[0]) * (c[0] - b[0]) + (c[1] - b[1]) * (c[1] - b[1]) + (c[2] - b[2]) * (c[2] - b[2]));

			//float angle = calc_angle(rj, ri, rk);
			float angle = calc_angle_abcb(drji, drki);

			//float cos_1 = calc_cos_angle(rj, ri, rk); // ji, ki
			float cos_1 = calc_cos_angle_abcb(drji, drki); // ji, ki
			//float cos_2 = calc_cos_angle(rj, rk, ri); // jk, ik
			float cos_2 = calc_cos_angle_abcb(drjk, drik); // jk, ik
			//float cos_3 = calc_cos_angle(ri, rj, rk); // ij, kj
			float cos_3 = calc_cos_angle_abcb(drij, drkj); // ij, kj

			float ksi3 = three_body_weight * (1.0 + 3 * cos_1 * cos_2 * cos_3) / powf(rij * rik * rjk, three_body_decay);

			float cos_angle = expf_v * 2.0 * cosf(angle);
			float sin_angle = expf_v * 2.0 * sinf(angle);

			int p = min(jelement, kelement);
			int q = max(jelement, kelement);

			int s = nelements * nRs2 + nRs3 * 2 * (-(p * (p + 1)) / 2 + q + nelements * p);

			for (int l = 0; l < nRs3; l++) {

				int z = s + l * 2;

				float radial = expf(-eta3 * powf(0.5 * (rij + rik) - sRs3[l], 2.0)) * rcutik * rcutij;

				atomicAdd(&output[molID][iatom][z], radial * cos_angle * ksi3);
				atomicAdd(&output[molID][iatom][z + 1], radial * sin_angle * ksi3);
			}
		}
	}
}

__global__ void fchl19_representation_and_derivative_cuda(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> charges,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> species,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> element_types,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> cell,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_cell,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockAtomIDs,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockMolIDs,
		const torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> neighbourlist,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> nneighbours, const int max_neighbours,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs2,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight,
		float three_body_decay, float rcut, torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> output,
		torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> grad) {

	extern __shared__ int s[];

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);

	int nelements = species.size(0);

	float *scoords_x = (float*) &s;
	float *scoords_y = (float*) &scoords_x[max_neighbours];
	float *scoords_z = (float*) &scoords_y[max_neighbours];
	int *selement_types = (int*) &scoords_z[max_neighbours];
	int *sneighbours = (int*) &selement_types[max_neighbours];

	float *sRs2 = (float*) &sneighbours[max_neighbours];
	float *sRs3 = (float*) &sRs2[nRs2];

	float *scell = (float*) &sRs3[nRs3];
	float *sinv_cell = (float*) &scell[9];

	int molID = blockMolIDs[blockIdx.x];
	int iatom = blockAtomIDs[blockIdx.x];
	int nneighbours_i = nneighbours[molID][iatom];

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		int j = neighbourlist[molID][iatom][jatom];

		scoords_x[jatom] = coordinates[molID][j][0];
		scoords_y[jatom] = coordinates[molID][j][1];
		scoords_z[jatom] = coordinates[molID][j][2];
		selement_types[jatom] = element_types[molID][j];
		sneighbours[jatom] = j;

	}

	for (int i = threadIdx.x; i < nRs2; i += blockDim.x) {
		sRs2[i] = Rs2[i];
	}

	for (int i = threadIdx.x; i < nRs3; i += blockDim.x) {
		sRs3[i] = Rs3[i];
	}

	bool pbc = false;

	if (cell.size(0) > 0) {

		pbc = true;

		if (threadIdx.x == 0 && threadIdx.y == 0) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					scell[i * 3 + j] = cell[molID][i][j];
					sinv_cell[i * 3 + j] = inv_cell[molID][i][j];
				}
			}
		}
	}

	__syncthreads();

	float ri[3];
	float rj[3];
	float rk[3];

	float drij[3];
	float drik[3];
	float drjk[3];

	ri[0] = coordinates[molID][iatom][0];
	ri[1] = coordinates[molID][iatom][1];
	ri[2] = coordinates[molID][iatom][2];

	float ielement = element_types[molID][iatom];

	float invcut = 1.0 / rcut;

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		rj[0] = scoords_x[jatom];
		rj[1] = scoords_y[jatom];
		rj[2] = scoords_z[jatom];

		int j = sneighbours[jatom];
		int jelement = selement_types[jatom];

		drij[0] = ri[0] - rj[0];
		drij[1] = ri[1] - rj[1];
		drij[2] = ri[2] - rj[2];

		if (pbc) {
			get_pbc_drij(drij, scell, sinv_cell);
		}

		float drji[3] = { -drij[0], -drij[1], -drij[2] };

		float rij2 = drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2];
		float rij = sqrtf(rij2);
		float invrij = 1.0 / rij;
		float invrij2 = invrij * invrij;

		float scaling = 1.0 / powf(rij, two_body_decay);

		float rcutij = get_cutoff(rij, rcut, 0.0, 0);

		float mu = log(rij / sqrt(1.0 + eta2 / powf(rij, 2.0)));
		float sigma = sqrt(log(1.0 + eta2 / powf(rij, 2.0)));

		float dcut = get_cutoff_derivative(rij, rcut, 0.0, 0);

		float dscal = -two_body_decay / powf(rij, two_body_decay + 1.0);

		for (int z = 0; z < nRs2; z++) {

			float radial = 1.0 / (sigma * sqrt(2.0 * M_PI) * sRs2[z]) * expf(-powf(log(sRs2[z]) - mu, 2) / (2.0 * powf(sigma, 2)));

			float rep = radial * scaling * rcutij;

			atomicAdd(&output[molID][iatom][jelement * nRs2 + z], rep);

			for (int x = 0; x < 3; x++) {

				float dx = drij[x] / rij;

				float dradialx = get_radial_derivative_distribution(dx, rij, eta2, sRs2, z, 1);

				float dcutx = dcut * -dx;

				float dscalingx = dscal * -dx;

				float deriv = dradialx * scaling * rcutij + radial * dscalingx * rcutij + radial * scaling * dcutx;

				atomicAdd(&grad[molID][iatom][iatom][x][jelement * nRs2 + z], -deriv);
				atomicAdd(&grad[molID][iatom][j][x][jelement * nRs2 + z], deriv);

			}

		}

		for (int katom = jatom + 1; katom < nneighbours_i; katom++) {

			rk[0] = scoords_x[katom];
			rk[1] = scoords_y[katom];
			rk[2] = scoords_z[katom];

			int kelement = selement_types[katom];
			int k = sneighbours[katom];

			drik[0] = ri[0] - rk[0];
			drik[1] = ri[1] - rk[1];
			drik[2] = ri[2] - rk[2];

			if (pbc) {
				get_pbc_drij(drik, scell, sinv_cell);
			}
			drjk[0] = drik[0] - drij[0];
			drjk[1] = drik[1] - drij[1];
			drjk[2] = drik[2] - drij[2];

			float drki[3] = { -drik[0], -drik[1], -drik[2] };
			float drkj[3] = { -drjk[0], -drjk[1], -drjk[2] };

			float rik2 = drik[0] * drik[0] + drik[1] * drik[1] + drik[2] * drik[2];
			float rik = sqrtf(rik2);

			float invrik = 1.0 / rik;
			float invrik2 = invrik * invrik;

			if (rik > rcut) {
				continue;
			}

			float rjk2 = drjk[0] * drjk[0] + drjk[1] * drjk[1] + drjk[2] * drjk[2];
			float rjk = sqrtf(rjk2);

			float invrjk = 1.0 / rjk;
			float invrjk2 = invrjk * invrjk;

			float rcutik = get_cutoff(rik, rcut, 0.0, 0);

			//float angle = calc_angle(rj, ri, rk);
			float angle = calc_angle_abcb(drji, drki);

			//float cos_i = calc_cos_angle(rj, ri, rk); // ji, ki
			float cos_i = calc_cos_angle_abcb(drji, drki); // ji, ki
			//float cos_k = calc_cos_angle(rj, rk, ri); // jk, ik
			float cos_k = calc_cos_angle_abcb(drjk, drik); // jk, ik
			//float cos_j = calc_cos_angle(ri, rj, rk); // ij, kj
			float cos_j = calc_cos_angle_abcb(drij, drkj); // ij, kj

			float cos_angle = expf(-powf(M_PI, 2) * 0.5) * 2.0 * cosf(angle);
			float sin_angle = expf(-powf(M_PI, 2) * 0.5) * 2.0 * sinf(angle);

			float invr_atm = powf(invrij * invrjk * invrik, three_body_decay);

			float atm = (1.0 + 3.0 * cos_i * cos_j * cos_k) * invr_atm * three_body_weight;

			int p = min(jelement, kelement);
			int q = max(jelement, kelement);

			int s = nelements * nRs2 + nRs3 * 2 * (-(p * (p + 1)) / 2 + q + nelements * p);

			for (int l = 0; l < nRs3; l++) {

				int z = s + l * 2;

				float radial = expf(-eta3 * powf(0.5 * (rij + rik) - sRs3[l], 2.0)) * rcutik * rcutij;

				atomicAdd(&output[molID][iatom][z], radial * cos_angle * atm);
				atomicAdd(&output[molID][iatom][z + 1], radial * sin_angle * atm);

			}

			//float vi = dot(rj, ri, rk, ri); // ji, ki
			float vi = dot_abcd(drji, drki);
			//float vj = dot(rk, rj, ri, rj); // kj, ij
			float vj = dot_abcd(drkj, drij);
			//float vk = dot(ri, rk, rj, rk); // ik, jk
			float vk = dot_abcd(drik, drjk);

			float dcos_angle = expf(-powf(M_PI, 2) * 0.5) * 2 * sinf(angle) / sqrt(max(1e-10, rij2 * rik2 - vi * vi));
			float dsin_angle = -expf(-powf(M_PI, 2) * 0.5) * 2 * cosf(angle) / sqrt(max(1e-10, rij2 * rik2 - vi * vi));

			float atm_i = (3.0 * cos_j * cos_k) * invr_atm * invrij * invrik;
			float atm_j = (3.0 * cos_k * cos_i) * invr_atm * invrij * invrjk;
			float atm_k = (3.0 * cos_i * cos_j) * invr_atm * invrjk * invrik;

			for (int x = 0; x < 3; x++) {

				float a = drji[x];
				float b = 0.0;
				float c = drki[x];

				float d_radial_d_j = (b - a) * invrij; // drij

				float d_radial_d_k = (b - c) * invrik;  // drik

				float d_radial_d_i = -(d_radial_d_j + d_radial_d_k);

				float d_angular_d_j = (c - b) + vi * ((b - a) * invrij2); // drki, drij

				float d_angular_d_k = (a - b) + vi * ((b - c) * invrik2); // drji, drik

				float d_angular_d_i = -(d_angular_d_j + d_angular_d_k);

				float d_ijdecay = -M_PI * (b - a) * sinf(M_PI * rij * invcut) * 0.5 * invrij * invcut; // drij

				float d_ikdecay = -M_PI * (b - c) * sinf(M_PI * rik * invcut) * 0.5 * invrik * invcut; // drik

				float d_atm_ii = 2 * b - a - c - vi * ((b - a) * invrij2 + (b - c) * invrik2); //- a - c = -drjk, drij, drik

				float d_atm_ij = c - a - vj * (b - a) * invrij2;				//drkj, drij

				float d_atm_ik = a - c - vk * (b - c) * invrik2;				//drjk, drik

				float d_atm_ji = c - b - vi * (a - b) * invrij2;				//drki, drji

				float d_atm_jj = 2 * a - b - c - vj * ((a - b) * invrij2 + (a - c) * invrjk2);

				float d_atm_jk = b - c - vk * (a - c) * invrjk2;

				float d_atm_ki = a - b - vi * (c - b) * invrik2;

				float d_atm_kj = b - a - vj * (c - a) * invrjk2;

				float d_atm_kk = 2 * c - a - b - vk * ((c - a) * invrjk2 + (c - b) * invrik2);

				float d_atm_extra_i = ((a - b) * invrij2 + (c - b) * invrik2) * atm * three_body_decay / three_body_weight;

				float d_atm_extra_j = ((b - a) * invrij2 + (c - a) * invrjk2) * atm * three_body_decay / three_body_weight;

				float d_atm_extra_k = ((a - c) * invrjk2 + (b - c) * invrik2) * atm * three_body_decay / three_body_weight;

				for (int l = 0; l < nRs3; l++) {

					float radial = expf(-eta3 * powf(0.5 * (rij + rik) - sRs3[l], 2.0));
					float d_radial = radial * eta3 * (0.5 * (rij + rik) - sRs3[l]);

					int z = s + l * 2;

					atomicAdd(&grad[molID][iatom][iatom][x][z],
							dcos_angle * d_angular_d_i * radial * atm * rcutij * rcutik + cos_angle * d_radial * d_radial_d_i * atm * rcutij * rcutik
									+ cos_angle * radial * (atm_i * d_atm_ii + atm_j * d_atm_ij + atm_k * d_atm_ik + d_atm_extra_i) * three_body_weight * rcutij
											* rcutik + cos_angle * radial * (d_ijdecay * rcutik + rcutij * d_ikdecay) * atm);

					atomicAdd(&grad[molID][iatom][iatom][x][z + 1],
							dsin_angle * d_angular_d_i * radial * atm * rcutij * rcutik + sin_angle * d_radial * d_radial_d_i * atm * rcutij * rcutik
									+ sin_angle * radial * (atm_i * d_atm_ii + atm_j * d_atm_ij + atm_k * d_atm_ik + d_atm_extra_i) * three_body_weight * rcutij
											* rcutik + sin_angle * radial * (d_ijdecay * rcutik + rcutij * d_ikdecay) * atm);

					atomicAdd(&grad[molID][iatom][j][x][z],
							dcos_angle * d_angular_d_j * radial * atm * rcutij * rcutik + cos_angle * d_radial * d_radial_d_j * atm * rcutij * rcutik
									+ cos_angle * radial * (atm_i * d_atm_ji + atm_j * d_atm_jj + atm_k * d_atm_jk + d_atm_extra_j) * three_body_weight * rcutij
											* rcutik - cos_angle * radial * d_ijdecay * rcutik * atm);

					atomicAdd(&grad[molID][iatom][j][x][z + 1],
							dsin_angle * d_angular_d_j * radial * atm * rcutij * rcutik + sin_angle * d_radial * d_radial_d_j * atm * rcutij * rcutik
									+ sin_angle * radial * (atm_i * d_atm_ji + atm_j * d_atm_jj + atm_k * d_atm_jk + d_atm_extra_j) * three_body_weight * rcutij
											* rcutik - sin_angle * radial * d_ijdecay * rcutik * atm);

					atomicAdd(&grad[molID][iatom][k][x][z],
							dcos_angle * d_angular_d_k * radial * atm * rcutij * rcutik + cos_angle * d_radial * d_radial_d_k * atm * rcutij * rcutik
									+ cos_angle * radial * (atm_i * d_atm_ki + atm_j * d_atm_kj + atm_k * d_atm_kk + d_atm_extra_k) * three_body_weight * rcutij
											* rcutik - cos_angle * radial * rcutij * d_ikdecay * atm);

					atomicAdd(&grad[molID][iatom][k][x][z + 1],
							dsin_angle * d_angular_d_k * radial * atm * rcutij * rcutik + sin_angle * d_radial * d_radial_d_k * atm * rcutij * rcutik
									+ sin_angle * radial * (atm_i * d_atm_ki + atm_j * d_atm_kj + atm_k * d_atm_kk + d_atm_extra_k) * three_body_weight * rcutij
											* rcutik - sin_angle * radial * rcutij * d_ikdecay * atm);

				}
			}
		}
	}
}

__global__ void fchl19_derivative_cuda(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> charges,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> species,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> element_types,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> cell,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_cell,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockAtomIDs, // blockIdx -> atom idx
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockMolIDs, // blockIdx -> molecule jdx
		const torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> neighbourlist,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> nneighbours, const int max_neighbours,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs2,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight,
		float three_body_decay, float rcut, torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> grad) {

	extern __shared__ int s[];

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);

	int nelements = species.size(0);

	float *scoords_x = (float*) &s;
	float *scoords_y = (float*) &scoords_x[max_neighbours];
	float *scoords_z = (float*) &scoords_y[max_neighbours];
	int *selement_types = (int*) &scoords_z[max_neighbours];
	int *sneighbours = (int*) &selement_types[max_neighbours];

	float *sRs2 = (float*) &sneighbours[max_neighbours];
	float *sRs3 = (float*) &sRs2[nRs2];

	float *scell = (float*) &sRs3[nRs3];
	float *sinv_cell = (float*) &scell[9];

	int molID = blockMolIDs[blockIdx.x];
	int iatom = blockAtomIDs[blockIdx.x];
	int nneighbours_i = nneighbours[molID][iatom];

	bool pbc = false;

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		int j = neighbourlist[molID][iatom][jatom];

		scoords_x[jatom] = coordinates[molID][j][0];
		scoords_y[jatom] = coordinates[molID][j][1];
		scoords_z[jatom] = coordinates[molID][j][2];
		selement_types[jatom] = element_types[molID][j];
		sneighbours[jatom] = j;

	}

	for (int i = threadIdx.x; i < nRs2; i += blockDim.x) {
		sRs2[i] = Rs2[i];
	}

	for (int i = threadIdx.x; i < nRs3; i += blockDim.x) {
		sRs3[i] = Rs3[i];
	}

	__syncthreads();

	if (cell.size(0) > 0) {

		pbc = true;

		if (threadIdx.x == 0 && threadIdx.y == 0) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					scell[i * 3 + j] = cell[molID][i][j];
					sinv_cell[i * 3 + j] = inv_cell[molID][i][j];
				}
			}
		}
	}

	__syncthreads();

	float ri[3];
	float rj[3];
	float rk[3];

	float drij[3];
	float drik[3];
	float drjk[3];

	ri[0] = coordinates[molID][iatom][0];
	ri[1] = coordinates[molID][iatom][1];
	ri[2] = coordinates[molID][iatom][2];

	float ielement = element_types[molID][iatom];

	float invcut = 1.0 / rcut;

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		rj[0] = scoords_x[jatom];
		rj[1] = scoords_y[jatom];
		rj[2] = scoords_z[jatom];

		int j = sneighbours[jatom];
		int jelement = selement_types[jatom];

		drij[0] = ri[0] - rj[0];
		drij[1] = ri[1] - rj[1];
		drij[2] = ri[2] - rj[2];

		if (pbc) {
			get_pbc_drij(drij, scell, sinv_cell);
		}

		float drji[3] = { -drij[0], -drij[1], -drij[2] };

		float rij2 = drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2];
		float rij = sqrtf(rij2);
		float invrij = 1.0 / rij;
		float invrij2 = invrij * invrij;

		float scaling = 1.0 / powf(rij, two_body_decay);

		float rcutij = get_cutoff(rij, rcut, 0.0, 0);

		float mu = log(rij / sqrt(1.0 + eta2 / powf(rij, 2.0)));
		float sigma = sqrt(log(1.0 + eta2 / powf(rij, 2.0)));

		float dcut = get_cutoff_derivative(rij, rcut, 0.0, 0);

		float dscal = -two_body_decay / powf(rij, two_body_decay + 1.0);

		for (int z = 0; z < nRs2; z++) {

			float radial = 1.0 / (sigma * sqrt(2.0 * M_PI) * sRs2[z]) * expf(-powf(log(sRs2[z]) - mu, 2) / (2.0 * powf(sigma, 2)));

			for (int x = 0; x < 3; x++) {

				float dx = drij[x] / rij;

				float dradialx = get_radial_derivative_distribution(dx, rij, eta2, sRs2, z, 1);

				float dcutx = dcut * -dx;

				float dscalingx = dscal * -dx;

				float deriv = dradialx * scaling * rcutij + radial * dscalingx * rcutij + radial * scaling * dcutx;

				atomicAdd(&grad[molID][iatom][iatom][x][jelement * nRs2 + z], -deriv);
				atomicAdd(&grad[molID][iatom][j][x][jelement * nRs2 + z], deriv);

			}

		}

		for (int katom = jatom + 1; katom < nneighbours_i; katom++) {

			rk[0] = scoords_x[katom];
			rk[1] = scoords_y[katom];
			rk[2] = scoords_z[katom];

			int kelement = selement_types[katom];
			int k = sneighbours[katom];

			drik[0] = ri[0] - rk[0];
			drik[1] = ri[1] - rk[1];
			drik[2] = ri[2] - rk[2];

			if (pbc) {
				get_pbc_drij(drik, scell, sinv_cell);
			}

			drjk[0] = drik[0] - drij[0];
			drjk[1] = drik[1] - drij[1];
			drjk[2] = drik[2] - drij[2];

			float rik2 = drik[0] * drik[0] + drik[1] * drik[1] + drik[2] * drik[2];
			float rik = sqrtf(rik2);

			float invrik = 1.0 / rik;
			float invrik2 = invrik * invrik;

			if (rik > rcut) {
				continue;
			}

			float rjk2 = drjk[0] * drjk[0] + drjk[1] * drjk[1] + drjk[2] * drjk[2];
			float rjk = sqrtf(rjk2);

			float invrjk = 1.0 / rjk;
			float invrjk2 = invrjk * invrjk;

			float rcutik = get_cutoff(rik, rcut, 0.0, 0);

			float drki[3] = { -drik[0], -drik[1], -drik[2] };
			float drkj[3] = { -drjk[0], -drjk[1], -drjk[2] };

			//float angle = calc_angle(rj, ri, rk);
			float angle = calc_angle_abcb(drji, drki);

			//float cos_i = calc_cos_angle(rj, ri, rk); // ji, ki
			float cos_i = calc_cos_angle_abcb(drji, drki); // ji, ki
			//float cos_k = calc_cos_angle(rj, rk, ri); // jk, ik
			float cos_k = calc_cos_angle_abcb(drjk, drik); // jk, ik
			//float cos_j = calc_cos_angle(ri, rj, rk); // ij, kj
			float cos_j = calc_cos_angle_abcb(drij, drkj); // ij, kj

			float cos_angle = expf(-powf(M_PI, 2) * 0.5) * 2.0 * cosf(angle);
			float sin_angle = expf(-powf(M_PI, 2) * 0.5) * 2.0 * sinf(angle);

			float invr_atm = powf(invrij * invrjk * invrik, three_body_decay);

			float atm = (1.0 + 3.0 * cos_i * cos_j * cos_k) * invr_atm * three_body_weight;

			int p = min(jelement, kelement);
			int q = max(jelement, kelement);

			int s = nelements * nRs2 + nRs3 * 2 * (-(p * (p + 1)) / 2 + q + nelements * p);

			//float vi = dot(rj, ri, rk, ri); // ji, ki
			float vi = dot_abcd(drji, drki);
			//float vj = dot(rk, rj, ri, rj); // kj, ij
			float vj = dot_abcd(drkj, drij);
			//float vk = dot(ri, rk, rj, rk); // ik, jk
			float vk = dot_abcd(drik, drjk);

			float dcos_angle = expf(-powf(M_PI, 2) * 0.5) * 2 * sinf(angle) / sqrt(max(1e-10, rij2 * rik2 - vi * vi));
			float dsin_angle = -expf(-powf(M_PI, 2) * 0.5) * 2 * cosf(angle) / sqrt(max(1e-10, rij2 * rik2 - vi * vi));

			float atm_i = (3.0 * cos_j * cos_k) * invr_atm * invrij * invrik;
			float atm_j = (3.0 * cos_k * cos_i) * invr_atm * invrij * invrjk;
			float atm_k = (3.0 * cos_i * cos_j) * invr_atm * invrjk * invrik;

			for (int x = 0; x < 3; x++) {

				float a = drji[x];
				float b = 0.0;
				float c = drki[x];

				float d_radial_d_j = (b - a) * invrij; // drij

				float d_radial_d_k = (b - c) * invrik;  // drik

				float d_radial_d_i = -(d_radial_d_j + d_radial_d_k);

				float d_angular_d_j = (c - b) + vi * ((b - a) * invrij2); // drki, drij

				float d_angular_d_k = (a - b) + vi * ((b - c) * invrik2); // drji, drik

				float d_angular_d_i = -(d_angular_d_j + d_angular_d_k);

				float d_ijdecay = -M_PI * (b - a) * sinf(M_PI * rij * invcut) * 0.5 * invrij * invcut; // drij

				float d_ikdecay = -M_PI * (b - c) * sinf(M_PI * rik * invcut) * 0.5 * invrik * invcut; // drik

				float d_atm_ii = 2 * b - a - c - vi * ((b - a) * invrij2 + (b - c) * invrik2); //- a - c = -drjk, drij, drik

				float d_atm_ij = c - a - vj * (b - a) * invrij2;				//drkj, drij

				float d_atm_ik = a - c - vk * (b - c) * invrik2;				//drjk, drik

				float d_atm_ji = c - b - vi * (a - b) * invrij2;				//drki, drji

				float d_atm_jj = 2 * a - b - c - vj * ((a - b) * invrij2 + (a - c) * invrjk2);

				float d_atm_jk = b - c - vk * (a - c) * invrjk2;

				float d_atm_ki = a - b - vi * (c - b) * invrik2;

				float d_atm_kj = b - a - vj * (c - a) * invrjk2;

				float d_atm_kk = 2 * c - a - b - vk * ((c - a) * invrjk2 + (c - b) * invrik2);

				float d_atm_extra_i = ((a - b) * invrij2 + (c - b) * invrik2) * atm * three_body_decay / three_body_weight;

				float d_atm_extra_j = ((b - a) * invrij2 + (c - a) * invrjk2) * atm * three_body_decay / three_body_weight;

				float d_atm_extra_k = ((a - c) * invrjk2 + (b - c) * invrik2) * atm * three_body_decay / three_body_weight;

				for (int l = 0; l < nRs3; l++) {

					float radial = expf(-eta3 * powf(0.5 * (rij + rik) - sRs3[l], 2.0));
					float d_radial = radial * eta3 * (0.5 * (rij + rik) - sRs3[l]);

					int z = s + l * 2;

					atomicAdd(&grad[molID][iatom][iatom][x][z],
							dcos_angle * d_angular_d_i * radial * atm * rcutij * rcutik + cos_angle * d_radial * d_radial_d_i * atm * rcutij * rcutik
									+ cos_angle * radial * (atm_i * d_atm_ii + atm_j * d_atm_ij + atm_k * d_atm_ik + d_atm_extra_i) * three_body_weight * rcutij
											* rcutik + cos_angle * radial * (d_ijdecay * rcutik + rcutij * d_ikdecay) * atm);

					atomicAdd(&grad[molID][iatom][iatom][x][z + 1],
							dsin_angle * d_angular_d_i * radial * atm * rcutij * rcutik + sin_angle * d_radial * d_radial_d_i * atm * rcutij * rcutik
									+ sin_angle * radial * (atm_i * d_atm_ii + atm_j * d_atm_ij + atm_k * d_atm_ik + d_atm_extra_i) * three_body_weight * rcutij
											* rcutik + sin_angle * radial * (d_ijdecay * rcutik + rcutij * d_ikdecay) * atm);

					atomicAdd(&grad[molID][iatom][j][x][z],
							dcos_angle * d_angular_d_j * radial * atm * rcutij * rcutik + cos_angle * d_radial * d_radial_d_j * atm * rcutij * rcutik
									+ cos_angle * radial * (atm_i * d_atm_ji + atm_j * d_atm_jj + atm_k * d_atm_jk + d_atm_extra_j) * three_body_weight * rcutij
											* rcutik - cos_angle * radial * d_ijdecay * rcutik * atm);

					atomicAdd(&grad[molID][iatom][j][x][z + 1],
							dsin_angle * d_angular_d_j * radial * atm * rcutij * rcutik + sin_angle * d_radial * d_radial_d_j * atm * rcutij * rcutik
									+ sin_angle * radial * (atm_i * d_atm_ji + atm_j * d_atm_jj + atm_k * d_atm_jk + d_atm_extra_j) * three_body_weight * rcutij
											* rcutik - sin_angle * radial * d_ijdecay * rcutik * atm);

					atomicAdd(&grad[molID][iatom][k][x][z],
							dcos_angle * d_angular_d_k * radial * atm * rcutij * rcutik + cos_angle * d_radial * d_radial_d_k * atm * rcutij * rcutik
									+ cos_angle * radial * (atm_i * d_atm_ki + atm_j * d_atm_kj + atm_k * d_atm_kk + d_atm_extra_k) * three_body_weight * rcutij
											* rcutik - cos_angle * radial * rcutij * d_ikdecay * atm);

					atomicAdd(&grad[molID][iatom][k][x][z + 1],
							dsin_angle * d_angular_d_k * radial * atm * rcutij * rcutik + sin_angle * d_radial * d_radial_d_k * atm * rcutij * rcutik
									+ sin_angle * radial * (atm_i * d_atm_ki + atm_j * d_atm_kj + atm_k * d_atm_kk + d_atm_extra_k) * three_body_weight * rcutij
											* rcutik - sin_angle * radial * rcutij * d_ikdecay * atm);

				}
			}
		}
	}
}

__global__ void fchl19_backwards_cuda(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> charges,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> species,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> element_types,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> cell,
		const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> inv_cell,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockAtomIDs, // blockIdx -> atom idx
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> blockMolIDs, // blockIdx -> molecule jdx
		const torch::PackedTensorAccessor32<int, 3, torch::RestrictPtrTraits> neighbourlist,
		const torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> nneighbours, const int max_neighbours,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs2,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight,
		float three_body_decay, float rcut, torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> grad_in,
		torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> grad_out) {

	extern __shared__ int s[];

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);

	int nelements = species.size(0);

	float *scoords_x = (float*) &s;
	float *scoords_y = (float*) &scoords_x[max_neighbours];
	float *scoords_z = (float*) &scoords_y[max_neighbours];
	int *selement_types = (int*) &scoords_z[max_neighbours];
	int *sneighbours = (int*) &selement_types[max_neighbours];

	float *sRs2 = (float*) &sneighbours[max_neighbours];
	float *sRs3 = (float*) &sRs2[nRs2];

	float *scell = (float*) &sRs3[nRs3];
	float *sinv_cell = (float*) &scell[9];

	float *sgradx = (float*) &sinv_cell[9];
	float *sgrady = (float*) &sgradx[max_neighbours];
	float *sgradz = (float*) &sgrady[max_neighbours];

	int molID = blockMolIDs[blockIdx.x];
	int iatom = blockAtomIDs[blockIdx.x];
	int nneighbours_i = nneighbours[molID][iatom];

	for (int jatom = threadIdx.y * blockDim.x + threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x * blockDim.y) {

		int j = neighbourlist[molID][iatom][jatom];

		scoords_x[jatom] = coordinates[molID][j][0];
		scoords_y[jatom] = coordinates[molID][j][1];
		scoords_z[jatom] = coordinates[molID][j][2];
		selement_types[jatom] = element_types[molID][j];
		sneighbours[jatom] = j;

	}
	__syncthreads();

	float igrad[3] = { 0.0, 0.0, 0.0 };

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < max_neighbours; i += blockDim.x * blockDim.y) {
		sgradx[i] = 0.0;
		sgrady[i] = 0.0;
		sgradz[i] = 0.0;
	}

	__syncthreads();

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nRs2; i += blockDim.x * blockDim.y) {
		sRs2[i] = Rs2[i];
	}

	__syncthreads();

	for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nRs3; i += blockDim.x * blockDim.y) {
		sRs3[i] = Rs3[i];
	}

	__syncthreads();

	bool pbc = false;

	if (cell.size(0) > 0) {

		pbc = true;

		if (threadIdx.x == 0 && threadIdx.y == 0) {
			for (int i = 0; i < 3; i++) {
				for (int j = 0; j < 3; j++) {
					scell[i * 3 + j] = cell[molID][i][j];
					sinv_cell[i * 3 + j] = inv_cell[molID][i][j];
				}
			}
		}
	}

	__syncthreads();

	float ri[3];
	float rj[3];
	float rk[3];

	float drij[3];
	float drik[3];
	float drjk[3];

	ri[0] = coordinates[molID][iatom][0];
	ri[1] = coordinates[molID][iatom][1];
	ri[2] = coordinates[molID][iatom][2];

	float ielement = element_types[molID][iatom];

	float invcut = 1.0 / rcut;

	float zeta_factor = expf(-powf(M_PI, 2) * 0.5);

	for (int jatom = threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x) {

		float deriv_jatom[3] = { 0.0, 0.0, 0.0 };

		rj[0] = scoords_x[jatom];
		rj[1] = scoords_y[jatom];
		rj[2] = scoords_z[jatom];

		int jelement = selement_types[jatom];

		drij[0] = ri[0] - rj[0];
		drij[1] = ri[1] - rj[1];
		drij[2] = ri[2] - rj[2];

		if (pbc) {
			get_pbc_drij(drij, scell, sinv_cell);
		}

		float drji[3] = { -drij[0], -drij[1], -drij[2] };

		float rij2 = drij[0] * drij[0] + drij[1] * drij[1] + drij[2] * drij[2];
		float rij = sqrtf(rij2);
		float invrij = 1.0 / rij;
		float invrij2 = invrij * invrij;

		float scaling = 1.0 / powf(rij, two_body_decay);

		float rcutij = get_cutoff(rij, rcut, 0.0, 0);

		float mu = log(rij / sqrt(1.0 + eta2 / powf(rij, 2.0)));
		float sigma = sqrt(log(1.0 + eta2 / powf(rij, 2.0)));

		float dcut = get_cutoff_derivative(rij, rcut, 0.0, 0);

		float dscal = -two_body_decay / powf(rij, two_body_decay + 1.0);

		for (int z = threadIdx.y; z < nRs2; z += blockDim.y) {

			int idx = jelement * nRs2 + z;

			float radial = 1.0 / (sigma * sqrt(2.0 * M_PI) * sRs2[z]) * expf(-powf(log(sRs2[z]) - mu, 2) / (2.0 * powf(sigma, 2)));

			float grad_in_iatom = grad_in[molID][iatom][idx];

			float deriv[3];

			for (int x = 0; x < 3; x++) {

				float dx = drij[x] / rij;

				float dradialx = get_radial_derivative_distribution(dx, rij, eta2, sRs2, z, 1);

				float dcutx = dcut * -dx;

				float dscalingx = dscal * -dx;

				deriv[x] = dradialx * scaling * rcutij + radial * dscalingx * rcutij + radial * scaling * dcutx;

			}

			igrad[0] += grad_in_iatom * -deriv[0];
			igrad[1] += grad_in_iatom * -deriv[1];
			igrad[2] += grad_in_iatom * -deriv[2];

			deriv_jatom[0] += grad_in_iatom * deriv[0];
			deriv_jatom[1] += grad_in_iatom * deriv[1];
			deriv_jatom[2] += grad_in_iatom * deriv[2];

		}

		for (int katom = jatom + 1 + threadIdx.y; katom < nneighbours_i; katom += blockDim.y) {

			rk[0] = scoords_x[katom];
			rk[1] = scoords_y[katom];
			rk[2] = scoords_z[katom];

			int kelement = selement_types[katom];

			drik[0] = ri[0] - rk[0];
			drik[1] = ri[1] - rk[1];
			drik[2] = ri[2] - rk[2];

			if (pbc) {
				get_pbc_drij(drik, scell, sinv_cell);
			}

			drjk[0] = drik[0] - drij[0];
			drjk[1] = drik[1] - drij[1];
			drjk[2] = drik[2] - drij[2];

			float rik2 = drik[0] * drik[0] + drik[1] * drik[1] + drik[2] * drik[2];
			float rik = sqrtf(rik2);

			float invrik = 1.0 / rik;
			float invrik2 = invrik * invrik;

			if (rik > rcut) {
				continue;
			}

			float rjk2 = drjk[0] * drjk[0] + drjk[1] * drjk[1] + drjk[2] * drjk[2];
			float rjk = sqrtf(rjk2);

			float invrjk = 1.0 / rjk;
			float invrjk2 = invrjk * invrjk;

			float rcutik = get_cutoff(rik, rcut, 0.0, 0);

			float drki[3] = { -drik[0], -drik[1], -drik[2] };
			float drkj[3] = { -drjk[0], -drjk[1], -drjk[2] };

			//float angle = calc_angle(rj, ri, rk);
			float angle = calc_angle_abcb(drji, drki);

			//float cos_i = calc_cos_angle(rj, ri, rk); // ji, ki
			float cos_i = calc_cos_angle_abcb(drji, drki); // ji, ki
			//float cos_k = calc_cos_angle(rj, rk, ri); // jk, ik
			float cos_k = calc_cos_angle_abcb(drjk, drik); // jk, ik
			//float cos_j = calc_cos_angle(ri, rj, rk); // ij, kj
			float cos_j = calc_cos_angle_abcb(drij, drkj); // ij, kj

			float cos_angle = zeta_factor * 2.0 * cosf(angle);
			float sin_angle = zeta_factor * 2.0 * sinf(angle);

			float invr_atm = powf(invrij * invrjk * invrik, three_body_decay);

			float atm = (1.0 + 3.0 * cos_i * cos_j * cos_k) * invr_atm * three_body_weight;

			int p = min(jelement, kelement);
			int q = max(jelement, kelement);

			int s = nelements * nRs2 + nRs3 * 2 * (-(p * (p + 1)) / 2 + q + nelements * p);

			//float vi = dot(rj, ri, rk, ri); // ji, ki
			float vi = dot_abcd(drji, drki);
			//float vj = dot(rk, rj, ri, rj); // kj, ij
			float vj = dot_abcd(drkj, drij);
			//float vk = dot(ri, rk, rj, rk); // ik, jk
			float vk = dot_abcd(drik, drjk);

			float dcos_angle = zeta_factor * 2.0 * sinf(angle) / sqrt(max(1e-10, rij2 * rik2 - vi * vi));
			float dsin_angle = -zeta_factor * 2.0 * cosf(angle) / sqrt(max(1e-10, rij2 * rik2 - vi * vi));

			float angular[2] = { cos_angle, sin_angle };
			float dangular[2] = { dcos_angle, dsin_angle };

			float atm_i = (3.0 * cos_j * cos_k) * invr_atm * invrij * invrik;
			float atm_j = (3.0 * cos_k * cos_i) * invr_atm * invrij * invrjk;
			float atm_k = (3.0 * cos_i * cos_j) * invr_atm * invrjk * invrik;

			float d_radial_d_j[3];
			float d_radial_d_k[3];
			float d_radial_d_i[3];

			float d_angular_d_j[3];
			float d_angular_d_k[3];
			float d_angular_d_i[3];

			float d_ijdecay[3];
			float d_ikdecay[3];

			float d_atm_ii[3];
			float d_atm_ij[3];
			float d_atm_ik[3];

			float d_atm_ji[3];
			float d_atm_jj[3];
			float d_atm_jk[3];

			float d_atm_ki[3];
			float d_atm_kj[3];
			float d_atm_kk[3];

			float d_atm_extra_i[3];
			float d_atm_extra_j[3];
			float d_atm_extra_k[3];

			for (int x = 0; x < 3; x++) {

				float a = drji[x];
				float b = 0.0;
				float c = drki[x];

				d_radial_d_j[x] = (b - a) * invrij;
				d_radial_d_k[x] = (b - c) * invrik;
				d_radial_d_i[x] = -(d_radial_d_j[x] + d_radial_d_k[x]);

				d_angular_d_j[x] = (c - b) + vi * ((b - a) * invrij2);
				d_angular_d_k[x] = (a - b) + vi * ((b - c) * invrik2);
				d_angular_d_i[x] = -(d_angular_d_j[x] + d_angular_d_k[x]);

				d_ijdecay[x] = -M_PI * (b - a) * sinf(M_PI * rij * invcut) * 0.5 * invrij * invcut;
				d_ikdecay[x] = -M_PI * (b - c) * sinf(M_PI * rik * invcut) * 0.5 * invrik * invcut;

				d_atm_ii[x] = 2 * b - a - c - vi * ((b - a) * invrij2 + (b - c) * invrik2);
				d_atm_ij[x] = c - a - vj * (b - a) * invrij2;
				d_atm_ik[x] = a - c - vk * (b - c) * invrik2;

				d_atm_ji[x] = c - b - vi * (a - b) * invrij2;
				d_atm_jj[x] = 2 * a - b - c - vj * ((a - b) * invrij2 + (a - c) * invrjk2);
				d_atm_jk[x] = b - c - vk * (a - c) * invrjk2;

				d_atm_ki[x] = a - b - vi * (c - b) * invrik2;
				d_atm_kj[x] = b - a - vj * (c - a) * invrjk2;
				d_atm_kk[x] = 2 * c - a - b - vk * ((c - a) * invrjk2 + (c - b) * invrik2);

				d_atm_extra_i[x] = ((a - b) * invrij2 + (c - b) * invrik2) * atm * three_body_decay / three_body_weight;
				d_atm_extra_j[x] = ((b - a) * invrij2 + (c - a) * invrjk2) * atm * three_body_decay / three_body_weight;
				d_atm_extra_k[x] = ((a - c) * invrjk2 + (b - c) * invrik2) * atm * three_body_decay / three_body_weight;
			}

			float deriv_katom[3] = { 0.0, 0.0, 0.0 };

			float rcuts = rcutij * rcutik;

			float atm_cut = atm * rcuts;

			for (int l = 0; l < nRs3; l += 1) {

				float radial = expf(-eta3 * powf(0.5 * (rij + rik) - sRs3[l], 2.0));
				float d_radial = radial * eta3 * (0.5 * (rij + rik) - sRs3[l]);

				float grad_in_iatom_list[2] = { grad_in[molID][iatom][s + l * 2], grad_in[molID][iatom][s + l * 2 + 1] };

				for (int x = 0; x < 3; x++) {

					for (int a = 0; a < 2; a++) {

						float dangular_a = dangular[a];
						float angular_a = angular[a];

						float grad_in_iatom = grad_in_iatom_list[a];

						float deriv_iatomx = dangular_a * d_angular_d_i[x] * radial * atm_cut + angular_a * d_radial * d_radial_d_i[x] * atm_cut
								+ angular_a * radial * (atm_i * d_atm_ii[x] + atm_j * d_atm_ij[x] + atm_k * d_atm_ik[x] + d_atm_extra_i[x]) * three_body_weight
										* rcuts + angular_a * radial * (d_ijdecay[x] * rcutik + rcutij * d_ikdecay[x]) * atm;

						float deriv_jatomx = dangular_a * d_angular_d_j[x] * radial * atm_cut + angular_a * d_radial * d_radial_d_j[x] * atm_cut
								+ angular_a * radial * (atm_i * d_atm_ji[x] + atm_j * d_atm_jj[x] + atm_k * d_atm_jk[x] + d_atm_extra_j[x]) * three_body_weight
										* rcuts - angular_a * radial * d_ijdecay[x] * rcutik * atm;

						float deriv_katomx = dangular_a * d_angular_d_k[x] * radial * atm_cut + angular_a * d_radial * d_radial_d_k[x] * atm_cut
								+ angular_a * radial * (atm_i * d_atm_ki[x] + atm_j * d_atm_kj[x] + atm_k * d_atm_kk[x] + d_atm_extra_k[x]) * three_body_weight
										* rcuts - angular_a * radial * rcutij * d_ikdecay[x] * atm;

						igrad[x] += grad_in_iatom * deriv_iatomx;
						deriv_jatom[x] += grad_in_iatom * deriv_jatomx;
						deriv_katom[x] += grad_in_iatom * deriv_katomx;

					}
				}
			}

			atomicAdd(&sgradx[katom], deriv_katom[0]);
			atomicAdd(&sgrady[katom], deriv_katom[1]);
			atomicAdd(&sgradz[katom], deriv_katom[2]);
		}

		atomicAdd(&sgradx[jatom], deriv_jatom[0]);
		atomicAdd(&sgrady[jatom], deriv_jatom[1]);
		atomicAdd(&sgradz[jatom], deriv_jatom[2]);
	}

	__syncthreads();

	atomicAdd(&grad_out[molID][iatom][0], igrad[0]);
	atomicAdd(&grad_out[molID][iatom][1], igrad[1]);
	atomicAdd(&grad_out[molID][iatom][2], igrad[2]);

	for (int jatom = threadIdx.y * blockDim.x + threadIdx.x; jatom < nneighbours_i; jatom += blockDim.x * blockDim.y) {
		int j = sneighbours[jatom];

		atomicAdd(&grad_out[molID][j][0], sgradx[jatom]);
		atomicAdd(&grad_out[molID][j][1], sgrady[jatom]);
		atomicAdd(&grad_out[molID][j][2], sgradz[jatom]);
	}
}

__global__
void get_element_types_kernel(const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> coordinates,
		const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> charges,
		const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> natom_counts,
		const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> species,
		torch::PackedTensorAccessor32<int, 2, torch::RestrictPtrTraits> element_types) {

	int natoms = natom_counts[blockIdx.x];
	int nspecies = species.size(0);

	for (int iatom = threadIdx.x; iatom < natoms; iatom += blockDim.x) {

		if (iatom < natoms) {

			int qi = charges[blockIdx.x][iatom];

			int index = -1;
			for (int j = 0; j < nspecies; j++) {
				if (qi == species[j]) {
					index = j;
				}
			}

			element_types[blockIdx.x][iatom] = index;
		}
	}
}

void getElementTypesCUDA(torch::Tensor coordinates, torch::Tensor charges, torch::Tensor natom_counts, torch::Tensor species, torch::Tensor element_types) {

	int nbatch = coordinates.size(0);
	const int nthreads = 32;

	get_element_types_kernel<<<nbatch, nthreads>>>(coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			charges.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			natom_counts.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			species.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			element_types.packed_accessor32<int, 2, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}

void FCHLCuda_old(torch::Tensor coordinates, torch::Tensor charges, torch::Tensor species, torch::Tensor element_types, torch::Tensor cell,
		torch::Tensor inv_cell, torch::Tensor blockAtomIDs, torch::Tensor blockMolIDs, torch::Tensor neighbourlist, torch::Tensor nneighbours,
		torch::Tensor Rs2, torch::Tensor Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight, float three_body_decay, float rcut,
		torch::Tensor output) {

	const int nthreadsx = 32;
	const int nthreadsy = 1;

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);
	int nspecies = species.size(0);

//int repsize = nspecies * nRs2 + (nspecies * (nspecies + 1)) * nRs3;

	const int currBatch = blockAtomIDs.size(0);
	const int max_neighbours = nneighbours.max().item<int>();

	dim3 blocks(currBatch);

	dim3 grid(nthreadsx, nthreadsy);

	int shared_mem_size = nRs2 + nRs3 + 4 * max_neighbours + 18; //+ repsize;

	fchl19_representation_cuda<<<blocks, grid, shared_mem_size * sizeof(float)>>>(
			coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			charges.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			species.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			element_types.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
			cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			inv_cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			blockAtomIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			blockMolIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			neighbourlist.packed_accessor32<int, 3, torch::RestrictPtrTraits>(),
			nneighbours.packed_accessor32<int,2, torch::RestrictPtrTraits>(),
			max_neighbours,
			Rs2.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			Rs3.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			eta2, eta3, two_body_decay, three_body_weight, three_body_decay,rcut,

			output.packed_accessor32<float, 3, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}

void FCHLCuda(torch::Tensor coordinates, torch::Tensor charges, torch::Tensor species, torch::Tensor element_types, torch::Tensor cell, torch::Tensor inv_cell,
		torch::Tensor blockAtomIDs, torch::Tensor blockMolIDs, torch::Tensor neighbourlist, torch::Tensor nneighbours, torch::Tensor Rs2, torch::Tensor Rs3,
		float eta2, float eta3, float two_body_decay, float three_body_weight, float three_body_decay, float rcut, torch::Tensor output) {

	const int nthreadsx = 16;
	const int nthreadsy = 8;

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);
	int nspecies = species.size(0);

//int repsize = nspecies * nRs2 + (nspecies * (nspecies + 1)) * nRs3;

	const int currBatch = blockAtomIDs.size(0);
	const int max_neighbours = nneighbours.max().item<int>();

	dim3 blocks(currBatch);

	dim3 grid(nthreadsx, nthreadsy);

	int shared_mem_size = nRs2 + nRs3 + 4 * max_neighbours + 18; //+ repsize;

	fchl19_representation_cuda<<<blocks, grid, shared_mem_size * sizeof(float)>>>(
			coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			charges.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			species.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			element_types.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
			cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			inv_cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			blockAtomIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			blockMolIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			neighbourlist.packed_accessor32<int, 3, torch::RestrictPtrTraits>(),
			nneighbours.packed_accessor32<int,2, torch::RestrictPtrTraits>(),
			max_neighbours,
			Rs2.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			Rs3.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			eta2, eta3, two_body_decay, three_body_weight, three_body_decay,rcut,

			output.packed_accessor32<float, 3, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}

void FCHLDerivativeCuda(torch::Tensor coordinates, torch::Tensor charges, torch::Tensor species, torch::Tensor element_types, torch::Tensor cell,
		torch::Tensor inv_cell, torch::Tensor blockAtomIDs, torch::Tensor blockMolIDs, torch::Tensor neighbourlist, torch::Tensor nneighbours,
		torch::Tensor Rs2, torch::Tensor Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight, float three_body_decay, float rcut,
		torch::Tensor grad) {

	const int nthreads = 32;

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);
	int nspecies = species.size(0);

	const int currBatch = blockAtomIDs.size(0);
	const int max_neighbours = nneighbours.max().item<int>();

	int shared_mem_size = nRs2 + nRs3 + 5 * max_neighbours;

	fchl19_derivative_cuda<<<currBatch, nthreads, shared_mem_size * sizeof(float)>>>(
			coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			charges.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			species.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			element_types.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
			cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			inv_cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			blockAtomIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			blockMolIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			neighbourlist.packed_accessor32<int, 3, torch::RestrictPtrTraits>(),
			nneighbours.packed_accessor32<int,2, torch::RestrictPtrTraits>(),
			max_neighbours,
			Rs2.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			Rs3.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			eta2, eta3, two_body_decay, three_body_weight, three_body_decay,rcut,
			grad.packed_accessor32<float, 5, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}

void FCHLBackwardsCuda(torch::Tensor coordinates, torch::Tensor charges, torch::Tensor species, torch::Tensor element_types, torch::Tensor cell,
		torch::Tensor inv_cell, torch::Tensor blockAtomIDs, torch::Tensor blockMolIDs, torch::Tensor neighbourlist, torch::Tensor nneighbours,
		torch::Tensor Rs2, torch::Tensor Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight, float three_body_decay, float rcut,
		torch::Tensor grad_in, torch::Tensor grad_out) {

	const int nthreadsx = 16;
	const int nthreadsy = 8;

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);
	int nspecies = species.size(0);

	const int currBatch = blockAtomIDs.size(0);
	const int max_neighbours = nneighbours.max().item<int>();

	int shared_mem_size = nRs2 + nRs3 + 8 * max_neighbours;

	dim3 blocks(currBatch);

	dim3 grid(nthreadsx, nthreadsy);

	fchl19_backwards_cuda<<<blocks, grid, shared_mem_size * sizeof(float)>>>(
			coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			charges.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			species.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			element_types.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
			cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			inv_cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			blockAtomIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			blockMolIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			neighbourlist.packed_accessor32<int, 3, torch::RestrictPtrTraits>(),
			nneighbours.packed_accessor32<int,2, torch::RestrictPtrTraits>(),
			max_neighbours,
			Rs2.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			Rs3.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			eta2, eta3, two_body_decay, three_body_weight, three_body_decay,rcut,
			grad_in.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			grad_out.packed_accessor32<float, 3, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}

void FCHLRepresentationAndDerivativeCuda(torch::Tensor coordinates, torch::Tensor charges, torch::Tensor species, torch::Tensor element_types,
		torch::Tensor cell, torch::Tensor inv_cell, torch::Tensor blockAtomIDs, torch::Tensor blockMolIDs, torch::Tensor neighbourlist,
		torch::Tensor nneighbours, torch::Tensor Rs2, torch::Tensor Rs3, float eta2, float eta3, float two_body_decay, float three_body_weight,
		float three_body_decay, float rcut, torch::Tensor output, torch::Tensor grad) {

	const int nthreads = 32;

	int nRs2 = Rs2.size(0);
	int nRs3 = Rs3.size(0);
	int nspecies = species.size(0);

	const int currBatch = blockAtomIDs.size(0);
	const int max_neighbours = nneighbours.max().item<int>();

	int shared_mem_size = nRs2 + nRs3 + 5 * max_neighbours + 18;

	fchl19_representation_and_derivative_cuda<<<currBatch, nthreads, shared_mem_size * sizeof(float)>>>(
			coordinates.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			charges.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
			species.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			element_types.packed_accessor32<int, 2, torch::RestrictPtrTraits>(),
			cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			inv_cell.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
			blockAtomIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			blockMolIDs.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
			neighbourlist.packed_accessor32<int, 3, torch::RestrictPtrTraits>(),
			nneighbours.packed_accessor32<int,2, torch::RestrictPtrTraits>(),
			max_neighbours,
			Rs2.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			Rs3.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
			eta2, eta3, two_body_decay, three_body_weight, three_body_decay,rcut,

			output.packed_accessor32<float, 3, torch::RestrictPtrTraits>(), grad.packed_accessor32<float, 5, torch::RestrictPtrTraits>());

	hipDeviceSynchronize();

}
